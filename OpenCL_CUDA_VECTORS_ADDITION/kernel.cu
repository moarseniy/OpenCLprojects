#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

hipEvent_t start, stop;

__global__ void vectorAdd(float* a, float* b, float* c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	c[index] = a[index] + b[index];
}

void randomInit(float* a, int n)
{
	for (int i = 0; i < n; i++)
		a[i] = rand()%10;
}

int main(void)
{
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float gpuTime = 0.0;

	const unsigned int blockSize = 512;
	const unsigned int numBlocks = 3;
	const unsigned int numItems = numBlocks*blockSize;

	hipSetDevice(0);

	float* a = new float[numItems];
	float* b = new float[numItems];
	float* c = new float[numItems];

	randomInit(a, numItems);
	randomInit(b, numItems);

	float* aDev, * bDev, * cDev;

	hipMalloc((void**)&aDev, numItems * sizeof(float));
	hipMalloc((void**)&bDev, numItems * sizeof(float));
	hipMalloc((void**)&cDev, numItems * sizeof(float));

	hipMemcpy(aDev, a, numItems * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bDev, b, numItems * sizeof(float), hipMemcpyHostToDevice);


	hipEventRecord(start, 0);

	vectorAdd << <numBlocks, blockSize >> > (aDev, bDev, cDev);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("GPU time = %.4f \n", gpuTime);

	hipMemcpy((void*) c, cDev, numItems * sizeof(float), hipMemcpyDeviceToHost);

	

	delete[] a;
	delete[] b;
	delete[] c;

	hipFree(aDev);
	hipFree(bDev);
	hipFree(cDev);
}