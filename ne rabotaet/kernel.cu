#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

#define SIZE 256




hipEvent_t start, stop;

__global__ void sum_reduction(float* a, float* b)
{
	__shared__ float temp[SIZE]; 

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	temp[threadIdx.x] = a[id];
	
	__syncthreads();


	for (int i = 1; i < blockDim.x; i *= 2)
	{
		if (2*i* threadIdx.x <blockDim.x)
		{
			temp[2*i*threadIdx.x] += temp[2*i* threadIdx.x + i];
		}
		__syncthreads();
	}


	if (threadIdx.x == 0) 
	{
		b[blockIdx.x] = temp[0];
	}
}

void init_vector(float *a, int n) 
{
	for (int i = 0; i < n; i++) 
	{
		a[i] = 1.0;//rand() % 10;
	}
}

int main() 
{
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float gpuTime = 0.0;

	int n = 1 << 16;
	size_t bytes = n * sizeof(float);
	
	hipSetDevice(0);

	float *h_a, *h_b;
	float *d_a, *d_b;

	h_a = (float*)malloc(bytes);
	h_b= (float*)malloc(bytes);
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);

	init_vector(h_a, n);

	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

	int numBlocks = n / SIZE;

	hipEventRecord(start, 0);
	

	

	sum_reduction <<<numBlocks, SIZE >>> (d_a, d_b);

	


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("GPU time = %.4f \n", gpuTime);

	hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);

	float result = 0;
	for (int i = 0; i < numBlocks; i++)
	{
		result += h_b[i];
	}

	printf("Result= %f \n", result);


	free(h_a);
	free(h_b);
	hipFree(d_a);
	hipFree(d_b);

	return 0;
}

